#include "hip/hip_runtime.h"
//
//	Copyright :
//		Don't use this in commercial code unless you talk to me and I agree after stipulations.
//
// Description :
//		Iterative Two-Opt solver.
//
//	Author :
//		Name	: Jeffrey A Robinson
//		Email	: jarobinson3@crimson.ua.edu
//
//



// C++
#include <iostream>
#include <string>


// C
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>


// CUDA
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


// Force -Wall after this point, VC only (Check https://gcc.gnu.org/onlinedocs/gcc/Diagnostic-Pragmas.html for GCC)
#pragma warning(push,4)


// data structures
enum ThreadBufferStatus {MORE_THREADS_THAN_BUFFER,EQUAL_SIZE,MORE_BUFFER_THAN_THREADS};


// Data structure used to hold position along path
struct __align__(8) Data {
	float x,y;
};


// If not 0 then use Shared Memory Structure to hold x,y and w values; otherwise, each component is held in own array.
#define S_DATA 0

#if S_DATA == 1
// Data-structue for shared memory
struct __align__(8) S_Data {
	int w;
	float x,y;
};

#endif

// Global stats
static __device__ __managed__ int climbs_d = 0;
static __device__ __managed__ int best_d = INT_MAX;
static __device__ int restart_d = 0;




// Buffer space
#if S_DATA
extern __shared__ S_Data buffer[];
#else
extern __shared__ char buffer[];
__shared__ float *x_buffer;
__shared__ float *y_buffer;
__shared__ int   *w_buffer;
#endif



// Wrappers for the shared memory buffer(s)
static __device__ inline void sAtomicMinW(const int &index, const int &v) {
#if S_DATA
	atomicMin(&buffer[index].w,v);
#else
	atomicMin(w_buffer+index,v);
#endif
}

#if S_DATA
	#define sX(index,v) buffer[index].x = v
	#define sY(index,v) buffer[index].y = v
	#define sW(index,v) buffer[index].w = v
	
	#define gX(index) buffer[index].x
	#define gY(index) buffer[index].y
	#define gW(index) buffer[index].w
	
#else					
	#define sX(index,v) x_buffer[index] = v
	#define sY(index,v) y_buffer[index] = v
	#define sW(index,v) w_buffer[index] = v
	
	#define gX(index) x_buffer[index]
	#define gY(index) y_buffer[index]
	#define gW(index) w_buffer[index]
#endif


//
// Description :
// 	Give two points returns the distance between them
//
// @x1	- X value of the first point
// @x1	- Y value of the first point
// @x2	- X value of the second point
// @y2	- Y value of the second point
//
// @return - Returns the distance between the two points given
static __device__ inline float 
dist(float x1, float y1, float x2, float y2) {
	float x = x1-x2;
	float y = y1-y2; y *= y;
	return __float2int_rn(sqrtf(x*x + y));
}


//
// POSSIBLE IDEA :
// 		We could reduce the atomicAdd by letting each thread do their own work.
//		But when they run out then try to grab other blocks remaining work.
//
// Description :
// 		Returns a unique integer value with the intial value being 0
//
// @return  - Returns the next unique integer
static __device__ inline int 
nextInt() {
	if(threadIdx.x==0) {
		sW(0,atomicAdd(&restart_d,1));
	}__syncthreads();
	return gW(0);
}


//
// Description :
//		Allocates and initializes my global memory and shared memory.
//
//	@pos	- An array that need to be initialized and will hold our path points
//	@weight	- An array that need to be initialized and will hold our edge weights
//	@cities	- The amount of points in our graph
//
//	@return	- Returns true if initialization was successful, false otherwise.
template <int TileSize>
static inline __device__ bool
initMemory(const Data* &pos_d, Data* &pos, int * &weight, const int cities) {
	__shared__ Data *d;
	__shared__ int *w;
	// Allocate my global memory
	if(threadIdx.x == 0 ) {
		d = new Data[cities + 1];
		if(d != NULL) {
			w = new int[cities];
			if(w == NULL) {
				delete d;
				d = NULL;
			}
		}
	}__syncthreads();

	if(d == NULL) {
		return false;
	}

	// Save new memory locations
	pos = d;
	weight = w;

	for (int i = threadIdx.x; i < cities; i += blockDim.x) pos[i] = pos_d[i];
	__syncthreads();
#if S_DATA == 0
	// Initialize shared memory
	x_buffer = (float*)buffer;
	y_buffer = (float*)(buffer + sizeof(float) * TileSize);
	w_buffer = (int*)(buffer + 2 * sizeof(float) * TileSize);
#endif

	return true;
}



//
// Description :
// 		Each thread gives some integer value, then the "best" of them is returned.
//
// @t_val  - The number that the thread submits as a candidate for the maximum value
// @cities - The number of cities.
//
// @return - The best value of t_val seen from all threads
template <int Reductions,ThreadBufferStatus Status, int TileSize>
static inline __device__ int
maximum(int t_val, const int cities) {
	int upper = min(blockDim.x,min(TileSize,cities));
	
	if(Status == MORE_THREADS_THAN_BUFFER) {
		int Index = threadIdx.x % TileSize;
		w_buffer[Index] = t_val;
		__syncthreads();
		for(int i = 0 ; i <= (blockDim.x/TileSize); ++i ) {
			w_buffer[Index] = t_val = min(t_val,w_buffer[Index]);
		}
	}else {
		w_buffer[threadIdx.x] = t_val;
	}__syncthreads();
	
	// 1024
	if (TileSize > 512) {
		int offset = (upper + 1) / 2;	// 200
		if( threadIdx.x < offset) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x + offset]);
		}__syncthreads();
		upper = offset;
	}
	
	// 512
	if (TileSize > 256) {
		int offset = (upper + 1) / 2; // 100
		if( threadIdx.x < offset) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x + offset]);
		}__syncthreads();
		upper = offset;
	}
	
	// 256
	if (TileSize > 128) {
		int offset = (upper + 1) / 2; // 50
		if( threadIdx.x < offset) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x + offset]);
		}__syncthreads();
		upper = offset;
	}
	
	// 128
	if (TileSize > 64) {
		int offset = (upper + 1) / 2; // 25
		if( threadIdx.x < offset) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x + offset]);
		}__syncthreads();
		upper = offset;
	}
	
	// 64 and down
	if(threadIdx.x < 32) {
		if(TileSize > 32) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+(upper+1)/2]);
		}
		if(threadIdx.x < 16) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+16]);
		}
		if(threadIdx.x < 8) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+8]);
		}
		if(threadIdx.x < 4) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+4]);
		}
		if(threadIdx.x < 2) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+2]);
		}
		if(threadIdx.x < 1) {
			w_buffer[threadIdx.x] = t_val = min(t_val,w_buffer[threadIdx.x+1]);
		}
	}__syncthreads();
	
	return w_buffer[0];
}




//
// Description :
//		After we find the best four position to reconnect with all we need to
//		reverse the path between them.
//
//	@start 	 - The first position in the sub-path we have to swap with the end
// 	@end	 - The last position in the path we have to swap with the start
//	@pos	 - The positions in our path
//	@weights - The edge weights between points
static inline __device__ void
reverse(int start, int end, Data* &pos, int* &weight) {
	while(start<end) {

		int   w = weight[start];
		Data d = pos[start];

		weight[start] = weight[end-1];
		pos[start] = pos[end];

		weight[end-1] = w;
		pos[end] = d;

		start += blockDim.x;
		end -= blockDim.x;

	}__syncthreads();
}



//
// Description :
//		Perform a single iteration of Two-Opt.
//
// @pos			- The current Hamiltonian path
// @weight		- The current weight of our edges along the path
// @minchange 	- The current best change we can make
// @mini		- The ith city in the path that is part of the swap
// @minj		- The jth city in the path that is part of the swap
// @cities		- The number of cities along the path (excluding the end point)
template <ThreadBufferStatus Status,int TileSize>
static __device__ void
singleIter(Data* &pos, int* &weight, int &minchange, int &mini, int &minj, const int cities) {


	for (int ii = 0; ii < cities - 2; ii += blockDim.x) {
		int i = ii + threadIdx.x;
		float pxi0, pyi0, pxi1, pyi1, pxj1, pyj1;

		if (i < cities - 2) {
			minchange -= weight[i];
			pxi0 = pos[i].x;
			pyi0 = pos[i].y;
			pxi1 = pos[i+1].x;
			pyi1 = pos[i+1].y;
			pxj1 = pos[0].x;
			pyj1 = pos[0].y;
		}

		for (int jj = cities - 1; jj >= ii + 2; jj -= TileSize) {

			int bound = jj - TileSize + 1;

			if(Status==MORE_BUFFER_THAN_THREADS) {
				for(int k = threadIdx.x; k < TileSize; k += blockDim.x) {
					int index = k + bound;
					if (index >= (ii + 2)) {
						sX(k,pos[index].x);
						sY(k,pos[index].y);
						sW(k,weight[index]);
					}
				}
			}else {
				if(threadIdx.x < TileSize) {
					int index = threadIdx.x + bound;
					if (index >= (ii + 2)) {
						sX(threadIdx.x,pos[index].x);
						sY(threadIdx.x,pos[index].y);
						sW(threadIdx.x,weight[index]);
					}
				}
			}__syncthreads();

			int lower = bound;
			if (lower < i + 2) lower = i + 2;

			for (int j = jj; j >= lower; j--) {
				int jm = j - bound;

				float pxj0 = gX(jm);
				float pyj0 = gY(jm);
				int change = gW(jm) +
					+ dist(pxi0,pyi0,pxj0,pyj0)
					+ dist(pxi1,pyi1,pxj1,pyj1);

				pxj1 = pxj0;
				pyj1 = pyj0;

				if (minchange > change) {
					minchange = change;
					mini = i;
					minj = j;
				}
			}__syncthreads();
		}

		if (i < cities - 2) {
			minchange += weight[i];
		}
	}
}



//
// Description :
//		Perform the swaps to the edges i and j to decrease the total length of our
//		path and update the weight and pos arrays appropriately.
//
// @pos			- The current Hamiltonian path
// @weight		- The current weight of our edges along the path
// @minchange 	- The current best change we can make
// @mini		- The ith city in the path that is part of the swap
// @minj		- The jth city in the path that is part of the swap
// @cities		- The number of cities along the path (excluding the end point)
template <int Reductions, ThreadBufferStatus Status, int TileSize>
static __device__ bool
update(Data* &pos, int* &weight, int &minchange, int &mini, int &minj, const int cities) {

	//__shared__ int winner;winner = blockDim.x;
	if( maximum<Reductions,Status,TileSize>(minchange, cities) >= 0) return false;
	
	if(minchange == gW(0)) {
		sW(1,threadIdx.x);
	}__syncthreads();
	
	if(gW(1) == threadIdx.x) {
		sW(2,mini);
		sW(3,minj);
	}__syncthreads();
	
	mini = gW(2);
	minj = gW(3);

	// Fix path and weights
	reverse(mini+1+threadIdx.x,minj-threadIdx.x,pos,weight);

	// Fix connecting points
	weight[mini] = -dist(pos[mini].x,pos[mini].y,pos[mini+1].x,pos[mini+1].y);
	weight[minj] = -dist(pos[minj].x,pos[minj].y,pos[minj+1].x,pos[minj+1].y);
	__syncthreads();
	return true;
}



//
// Description :
//		Given a path we randomly permute it into a new new path and then initialize 
//		the weights of the path.
//
// @pos			- The current Hamiltonian path
// @weight		- The current weight of our edges along the path
// @cities		- The number of cities along the path (excluding the end point)
static __device__ inline void
permute(Data* &pos, int* &weight, const int cities) {
	if (threadIdx.x == 0) {  // serial permutation
		hiprandState rndstate;
		hiprand_init(blockIdx.x, 0, 0, &rndstate);
		for (int i = 1; i < cities; i++) {
			int j = hiprand(&rndstate) % (cities - 1) + 1;

			Data d = pos[i];
			pos[i] = pos[j];
			pos[j] = d;
		}
		pos[cities] = pos[0];
	}__syncthreads();

	for (int i = threadIdx.x; i < cities; i += blockDim.x) weight[i] = -dist(pos[i].x, pos[i].y, pos[i+1].x, pos[i+1].y);
	__syncthreads();

}


//
// Releases memory and saves results
//
// @pos				- Pointer to allocated path memory
// @weight			- Pointer to allocated edge weight memory
// @local_climbs	- The number of climbs performed by this block
// @best_length		- The best length this block found.
static __device__ void inline
cleanup(Data* &pos, int* &weight, int &local_climbs, int &best_length) {
	if (threadIdx.x == 0) {
		// Save data
		atomicAdd(&climbs_d,local_climbs);
		atomicMin(&best_d, best_length);
		
		// Release memory
		delete pos;
		delete weight;
	}
}

//
// Description :
//		Perform iterative two-opt until there can be no more swaps to reduce
//		the path length.
//
// @pos_d	- The position of each point in the graph.
// @cities	- The number of vertices in the graph
template <int Reductions,ThreadBufferStatus Status, int TileSize>
static __global__ __launch_bounds__(1024, 2) void
TwoOpt(const int Restarts, const Data *pos_d, const int cities) {

	Data	*pos;
	int 	*weight;
	int 	local_climbs = 0;
	int		best_length = INT_MAX;

	if( !initMemory<TileSize>(pos_d,pos,weight,cities) ) {
		if(threadIdx.x == 0) {
			printf("Memory initialization error for block %d\n", blockIdx.x);
		}
		return;
	}


	for(int r = nextInt() ; r < Restarts; r = nextInt()) {

		int mini,minj,minchange;

		permute(pos,weight,cities);

		do {
			++local_climbs;					// Global statistics
			minchange = mini = minj = 0;	// Reset
			singleIter<Status,TileSize>(pos, weight, minchange, mini, minj, cities);
		} while (update<Reductions,Status,TileSize>(pos, weight, minchange, mini, minj, cities));

		__shared__ int w; w = 0;
		__syncthreads();
		int term = 0;
		for (int i = threadIdx.x; i < cities; i += blockDim.x) {
			term += dist(pos[i].x, pos[i].y, pos[i+1].x, pos[i+1].y);
		}
		atomicAdd(&w,term);
		__syncthreads();

		if(threadIdx.x==0) {
			if(w < best_length) {
				best_length = w;
			}
		}

	}
	cleanup(pos, weight, local_climbs, best_length);
}



//
// Description :
//		Checks to see if an error occured with CUDA and if so prints out the message 
//		passed and the CUDA
// error then quits the application.
//
// @msg	- Message to print out if error occurs
static void
CudaTest(const char *msg) {
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
	system("PAUSE");
    exit(-1);
  }
}



// Terrible (TODO: Turn into functions)
#define mallocOnGPU(addr, size) if (hipSuccess != hipMalloc((void **)&addr, size)) fprintf(stderr, "could not allocate GPU memory\n");  CudaTest("couldn't allocate GPU memory");
#define copyToGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyHostToDevice)) fprintf(stderr, "copying of data to device failed\n");  CudaTest("data copy to device failed");



//
// Description :
// 		Read TPS lib files into GPU memory.  ATT and CEIL_2D edge weight types are 
//		not supported
//
// @fname	- The name of the file to read the TSP data from
// @pos_d	- Pointer to the pointer that will hold data on GPU
//			  and is modified here to be the address on the GPU
//
// @return	- Returns the number of cities found
static int
readInput(const char *fname, Data **pos_d) {
  int ch, cnt, in1, cities;
  float in2, in3;
  FILE *f;
  Data *pos;
  char str[256];  // potential for buffer overrun

  f = fopen(fname, "rt");
  if (f == NULL) {fprintf(stderr, "could not open file %s\n", fname);  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

  ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
  fscanf(f, "%s\n", str);
  cities = atoi(str);
  if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

  pos = new Data[cities];  if (pos == NULL) {fprintf(stderr, "cannot allocate pos\n");  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  fscanf(f, "%s\n", str);
  if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

  cnt = 0;
  while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) {

    pos[cnt].x = in2;
    pos[cnt].y = in3;

	++cnt;

    if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
    if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
  }
  if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}

  fscanf(f, "%s", str);
  if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}

  mallocOnGPU(*pos_d, sizeof(Data) * cities);
  copyToGPU(*pos_d, pos, sizeof(Data) * cities);

  fclose(f);

  delete (pos);

  return cities;
}



//
// Description :
// 		Given an enum value return it's string representation
//
// @status - The enum value to translate
//
// @return - The enums string representation in the source code
static const std::string
getName(const ThreadBufferStatus status) {
	switch(status) {
		case MORE_THREADS_THAN_BUFFER:
			return std::string("MORE_THREADS_THAN_BUFFER");
		case EQUAL_SIZE:
			return std::string("EQUAL_SIZE");
		case MORE_BUFFER_THAN_THREADS:
			return std::string("MORE_BUFFER_THAN_THREADS");
	};
	return std::string("enum value not found.");
}



//
// Description :
// 		Given an integer returns the next multiple of 32 greater than or equal to it.
//
// @in 		- The integer to round to next multiple of 32
//
// @return 	- Returns the next multiple of 32 that is greater than or equals to in
static int
next32(int in) {
	return ((in + 31) / 32 ) * 32;
}



//
// Description :
//		How many reductions do we need to perform in order to make sure we have found
//		our min/max/etc
//
// @return returns the number of reductions needed to propogate any value
static int
computeReductions(const int Cities, const int Threads, const int TileSize) {
	int MaxData = min(Threads,min(TileSize,Cities));
	if(MaxData>512) return 10;
	if(MaxData>256) return 9;
	if(MaxData>128) return 8;
	if(MaxData>64) return 7;
	if(MaxData>32) return 6;
	return 5;
}



//
// Description :
// Calculates the maximum number of resident blocks that the card can hold
//
// @Threads 		- Number of threads that each block will have
// @Shared_Bytes	- The amount of bytes each block will allocate
//
// @return 			- Returns the number of blocks the card can have resident
static int
getMaxBlocks(const int Shared_Bytes, const int Threads) {
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props,0);

	if(props.major < 3) {
		const int Max_Shared = 16384;
		const int Block_Shared_Limit = (Max_Shared / Shared_Bytes);
		return props.multiProcessorCount * min(8,min(Block_Shared_Limit,(int)(2048/Threads)));
	}else if(props.major < 5) {
		const int Max_Shared = 32768;
		const int Block_Shared_Limit = (Max_Shared / Shared_Bytes);
		return props.multiProcessorCount * min(16,min(Block_Shared_Limit,(int)(2048/Threads)));
	}else {
		const int Max_Shared = 65536;
		const int Block_Shared_Limit = (Max_Shared / Shared_Bytes);
		return props.multiProcessorCount * min(32,min(Block_Shared_Limit,(int)(2048/Threads)));
	}
}



//
// private : Handle ThreadBufferStatus kernel selection
//
template <int Reductions,int TileSize>
static float
_wrapStatus(const int Restarts, const int Threads, const Data *Pos_d, const int Cities) {

	float gpuExecutionTime;
	
	// A quick way to template out some checks.  If more threads than buffer then I have to atomicMin to a single slot.
	const ThreadBufferStatus Status = (Threads > TileSize) ? MORE_THREADS_THAN_BUFFER : (Threads < TileSize) ? MORE_BUFFER_THAN_THREADS : EQUAL_SIZE;
	
	// Amount of shared memory in Bytes
#if S_DATA
	const int Shared_Bytes = sizeof(S_Data) * TileSize;
#else
	const int Shared_Bytes = (sizeof(int) + 2 * sizeof(float)) * TileSize;
#endif

	// Calculate number of maximum number of resident blocks allowed on the card
	const int Blocks = min(Restarts,getMaxBlocks(Shared_Bytes,Threads));

	// Output runtime configuration
	std::cout	<< "Blocks = " << Blocks 
				<< ", Threads  = " << Threads 
				<< ", TileSize = " << TileSize 
				<< ", Status = " << getName(Status) 
				<< ", Reductions = " << Reductions 
				<< ", Shared Bytes = " << Shared_Bytes << std::endl;

	hipEvent_t begin,end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	switch(Status) {
		case MORE_THREADS_THAN_BUFFER:
			hipEventRecord(begin,0);
			TwoOpt<Reductions,MORE_THREADS_THAN_BUFFER,TileSize><<<Blocks,Threads,Shared_Bytes>>>(Restarts,Pos_d,Cities);
			hipEventRecord(end,0);
			hipEventSynchronize(end);
			break;
		case EQUAL_SIZE:
			hipEventRecord(begin,0);
			TwoOpt<Reductions,EQUAL_SIZE,TileSize><<<Blocks,Threads,Shared_Bytes>>>(Restarts,Pos_d,Cities);
			hipEventRecord(end,0);
			hipEventSynchronize(end);
			break;
		case MORE_BUFFER_THAN_THREADS:
			hipEventRecord(begin,0);
			TwoOpt<Reductions,MORE_BUFFER_THAN_THREADS,TileSize><<<Blocks,Threads,Shared_Bytes>>>(Restarts,Pos_d,Cities);
			hipEventRecord(end,0);
			hipEventSynchronize(end);
			break;
	};
	
	hipEventElapsedTime(&gpuExecutionTime,begin,end);

	hipEventDestroy(begin);
	hipEventDestroy(end);

	return gpuExecutionTime;
}



//
// private : Handle Reduction kernel selection
//
template <int TileSize>
static float
_wrapReduction(const int Restarts, const int Threads, const Data *Pos, const int Cities) {
	const int Reductions = computeReductions(Cities,Threads,TileSize);
	switch(Reductions) {
		case 10:
			return _wrapStatus<10,TileSize>(Restarts, Threads, Pos, Cities);
		case 9:
			return _wrapStatus<9,TileSize>(Restarts, Threads, Pos, Cities);
		case 8:
			return _wrapStatus<8,TileSize>(Restarts, Threads, Pos, Cities);
		case 7:
			return _wrapStatus<7,TileSize>(Restarts, Threads, Pos, Cities);
		case 6:
			return _wrapStatus<6,TileSize>(Restarts, Threads, Pos, Cities);
		default:
			return _wrapStatus<5,TileSize>(Restarts, Threads, Pos, Cities);
	}
}



//
// Description :
//		Given these parameters we construct and start a CUDA kernel.
//
// @Cities		- Number of cities or nodes in the graph
// @Pos			- Position data of graph nodes.
// @Restarts	- How many different random permutations of input city should be try
// @Threads		- The number of threads each block should have
// @TileSize	- The shared buffer size for our sliding tile.
//
// @return		- Returns the duration of the kernel in milliseconds.
static float
RunKernel(const int Cities, const Data *Pos, const int Restarts, const int Threads, const int TileSize) {
	switch(TileSize) {
		case 32:
			return _wrapReduction<32>(Restarts,Threads,Pos,Cities);
		case 64:
			return _wrapReduction<64>(Restarts,Threads,Pos,Cities);
		case 96:
			return _wrapReduction<96>(Restarts,Threads,Pos,Cities);
		case 128:
			return _wrapReduction<128>(Restarts,Threads,Pos,Cities);
		case 160:
			return _wrapReduction<160>(Restarts,Threads,Pos,Cities);
		case 192:
			return _wrapReduction<192>(Restarts,Threads,Pos,Cities);
		case 224:
			return _wrapReduction<224>(Restarts,Threads,Pos,Cities);
		case 256:
			return _wrapReduction<256>(Restarts,Threads,Pos,Cities);
		case 288:
			return _wrapReduction<288>(Restarts,Threads,Pos,Cities);
		case 320:
			return _wrapReduction<320>(Restarts,Threads,Pos,Cities);
		case 352:
			return _wrapReduction<352>(Restarts,Threads,Pos,Cities);
		case 384:
			return _wrapReduction<384>(Restarts,Threads,Pos,Cities);
		case 416:
			return _wrapReduction<416>(Restarts,Threads,Pos,Cities);
		case 448:
			return _wrapReduction<448>(Restarts,Threads,Pos,Cities);
		case 480:
			return _wrapReduction<480>(Restarts,Threads,Pos,Cities);
		case 512:
			return _wrapReduction<512>(Restarts,Threads,Pos,Cities);
		case 544:
			return _wrapReduction<544>(Restarts,Threads,Pos,Cities);
		case 576:
			return _wrapReduction<576>(Restarts,Threads,Pos,Cities);
		case 608:
			return _wrapReduction<608>(Restarts,Threads,Pos,Cities);
		case 640:
			return _wrapReduction<640>(Restarts,Threads,Pos,Cities);
		case 672:
			return _wrapReduction<672>(Restarts,Threads,Pos,Cities);
		case 704:
			return _wrapReduction<704>(Restarts,Threads,Pos,Cities);
		case 736:
			return _wrapReduction<736>(Restarts,Threads,Pos,Cities);
		case 768:
			return _wrapReduction<768>(Restarts,Threads,Pos,Cities);
		case 800:
			return _wrapReduction<800>(Restarts,Threads,Pos,Cities);
		case 832:
			return _wrapReduction<832>(Restarts,Threads,Pos,Cities);
		case 864:
			return _wrapReduction<864>(Restarts,Threads,Pos,Cities);
		case 896:
			return _wrapReduction<896>(Restarts,Threads,Pos,Cities);
		case 928:
			return _wrapReduction<928>(Restarts,Threads,Pos,Cities);
		case 960:
			return _wrapReduction<960>(Restarts,Threads,Pos,Cities);
		case 992:
			return _wrapReduction<992>(Restarts,Threads,Pos,Cities);
		case 1024:
			return _wrapReduction<1024>(Restarts,Threads,Pos,Cities);
		default:
			std::cout << "Invalid TileSize = " << TileSize << std::endl;
			exit(-1);
	};
	return -1;
}



//
// Description :
//		Main entry point to our iterative Two-Opt solver.  
//		Options are ./<name> problem_file restarts <threads> <buffer_size>
//
// @argc	- Number of command line parameters (including program name)
// @argv	- Holds command line arguments
//
// @return	- Returns 0 if success, otherwise failure.
int
main(int argc, char *argv[]) {
	printf("2-opt TSP CUDA GPU code v2.1 [Kepler]\n");
	printf("Copyright (c) 2014, Texas State University. All rights reserved.\n");

	if (argc < 3 || argc > 5) {fprintf(stderr, "\narguments: input_file restart_count <threads> <tilesize> \n"); exit(-1);}

	const int Restarts = atoi(argv[2]);
	if (Restarts < 1) {fprintf(stderr, "restart_count is too small: %d\n", Restarts); exit(-1);}

	Data *pos_d;
	const int Cities = readInput(argv[1], &pos_d);
	printf("configuration: %d cities, %d restarts, %s input\n", Cities, Restarts, argv[1]);

	const int Threads = min(1024,(argc >= 4) ? next32(atoi(argv[3])) : next32(Cities));
	const int TileSize = min(1024,(argc >= 5) ? next32(atoi(argv[4])) : Threads);

	const float GpuExecutionTime = RunKernel(Cities,pos_d,Restarts,Threads,TileSize);

	int hours = (int)(GpuExecutionTime / (3600.0f * 1000.0f));
	int seconds = (int)(GpuExecutionTime/1000) % 60;
	int minutes = (int)(GpuExecutionTime/1000) / 60;

	long long moves = 1LL * climbs_d * (Cities - 2) * (Cities - 1) / 2;

	std::cout << moves * 0.000001 / GpuExecutionTime << "Gmoves/s" << std::endl;
	std::cout << "best found tour length = " << best_d << std::endl;
	std::cout << "Total Time : " << GpuExecutionTime / 1000.0f << "s" << std::endl;
	std::cout << "Hours = " << hours << ", Minutes = " << minutes << ", Seconds = " << seconds << ", Milliseconds = " << (int)(GpuExecutionTime) % 1000 << std::endl;

	hipDeviceReset();
	hipFree(pos_d);
	return 0;
}

